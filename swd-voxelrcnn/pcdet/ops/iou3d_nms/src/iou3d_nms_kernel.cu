#include "hip/hip_runtime.h"
/*
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/


#include <stdio.h>
#define THREADS_PER_BLOCK 16
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

// #define DEBUG
const int THREADS_PER_BLOCK_NMS = sizeof(unsigned long long) * 8;
const float EPS = 1e-8;
struct Point {
    float y, z;
    __device__ Point() {}
    __device__ Point(double _x, double _y){
        y = _y, z = _z;
    }

    __device__ void set(float _x, float _y){
        y = _y; z = _z;
    }

    __device__ Point operator +(const Point &b)const{
        return Point(y + b.y, z + b.z);
    }

    __device__ Point operator -(const Point &b)const{
        return Point(y - b.y, z - b.z);
    }
};

__device__ inline float cross(const Point &a, const Point &b){
    return a.y * b.z - a.z * b.y;
}

__device__ inline float cross(const Point &p1, const Point &p2, const Point &p0){
    return (p1.y - p0.y) * (p2.z - p0.z) - (p2.y - p0.y) * (p1.z - p0.z);
}

__device__ int check_rect_cross(const Point &p1, const Point &p2, const Point &q1, const Point &q2){
    int ret = min(p1.y,p2.y) <= max(q1.y,q2.y)  &&
              min(q1.y,q2.y) <= max(p1.y,p2.y) &&
              min(p1.z,p2.z) <= max(q1.z,q2.z) &&
              min(q1.z,q2.z) <= max(p1.z,p2.z);
    return ret;
}

__device__ inline int check_in_box2d(const float *box, const Point &p){
    //params: (7) [x, y, z, dx, dy, dz, heading]
    const float MARGIN = 1e-2;

    float center_y = box[1], center_z = box[2];
    float angle_cos = cos(-box[6]), angle_sin = sin(-box[6]);  // rotate the point in the opposite direction of box
    float rot_y = (p.y - center_y) * angle_cos + (p.z - center_z) * (-angle_sin);
    float rot_z = (p.y - center_y) * angle_sin + (p.z - center_z) * angle_cos;

    return (fabs(rot_y) < box[4] / 2 + MARGIN && fabs(rot_z) < box[5] / 2 + MARGIN);
}

__device__ inline int intersection(const Point &p1, const Point &p0, const Point &q1, const Point &q0, Point &ans){
    // fast exclusion
    if (check_rect_cross(p0, p1, q0, q1) == 0) return 0;

    // check cross standing
    float s1 = cross(q0, p1, p0);
    float s2 = cross(p1, q1, p0);
    float s3 = cross(p0, q1, q0);
    float s4 = cross(q1, p1, q0);

    if (!(s1 * s2 > 0 && s3 * s4 > 0)) return 0;

    // calculate intersection of two lines
    float s5 = cross(q1, p1, p0);
    if(fabs(s5 - s1) > EPS){
        ans.y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);
        ans.z = (s5 * q0.z - s1 * q1.z) / (s5 - s1);

    }
    else{
        float a0 = p0.z - p1.z, b0 = p1.y - p0.y, c0 = p0.y * p1.z - p1.y * p0.z;
        float a1 = q0.z - q1.z, b1 = q1.y - q0.y, c1 = q0.y * q1.z - q1.y * q0.z;
        float D = a0 * b1 - a1 * b0;

        ans.y = (b0 * c1 - b1 * c0) / D;
        ans.z = (a1 * c0 - a0 * c1) / D;
    }

    return 1;
}

__device__ inline void rotate_around_center(const Point &center, const float angle_cos, const float angle_sin, Point &p){
    float new_y = (p.y - center.y) * angle_cos + (p.z - center.z) * (-angle_sin) + center.y;
    float new_z = (p.y - center.y) * angle_sin + (p.z - center.z) * angle_cos + center.z;
    p.set(new_y, new_z);
}

__device__ inline int point_cmp(const Point &a, const Point &b, const Point &center){
    return -atan2(a.y - center.y, a.z - center.z) > -atan2(b.y - center.y, b.z - center.z);
}

__device__ inline float box_overlap(const float *box_a, const float *box_b){
    // params box_a: [x, y, z, dx, dy, dz, heading]
    // params box_b: [x, y, z, dx, dy, dz, heading]

    float a_angle = box_a[6], b_angle = box_b[6];
    float a_dy_half = box_a[4] / 2, b_dy_half = box_b[4] / 2, a_dz_half = box_a[5] / 2, b_dz_half = box_b[5] / 2;
    float a_y1 = box_a[1] - a_dy_half, a_z1 = box_a[2] - a_dz_half;
    float a_y2 = box_a[1] + a_dy_half, a_z2 = box_a[2] + a_dz_half;
    float b_y1 = box_b[1] - b_dy_half, b_z1 = box_b[2] - b_dz_half;
    float b_y2 = box_b[1] + b_dy_half, b_z2 = box_b[2] + b_dz_half;

    Point center_a(box_a[1], box_a[2]);
    Point center_b(box_b[1], box_b[2]);

#ifdef DEBUG
    printf("a: (%.3f, %.3f, %.3f, %.3f, %.3f), b: (%.3f, %.3f, %.3f, %.3f, %.3f)\n", a_y1, a_z1, a_y2, a_z2, a_angle,
           b_y1, b_z1, b_y2, b_z2, b_angle);
    printf("center a: (%.3f, %.3f), b: (%.3f, %.3f)\n", center_a.x, center_a.y, center_b.x, center_b.y);
#endif

    Point box_a_corners[5];
    box_a_corners[0].set(a_y1, a_z1);
    box_a_corners[1].set(a_y2, a_z1);
    box_a_corners[2].set(a_y2, a_z2);
    box_a_corners[3].set(a_y1, a_z2);

    Point box_b_corners[5];
    box_b_corners[0].set(b_y1, b_z1);
    box_b_corners[1].set(b_y2, b_z1);
    box_b_corners[2].set(b_y2, b_z2);
    box_b_corners[3].set(b_y1, b_z2);

    // get oriented corners
    float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
    float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

    for (int k = 0; k < 4; k++){
#ifdef DEBUG
        printf("before corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
        rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
        rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
#ifdef DEBUG
        printf("corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
    }

    box_a_corners[4] = box_a_corners[0];
    box_b_corners[4] = box_b_corners[0];

    // get intersection of lines
    Point cross_points[16];
    Point poly_center;
    int cnt = 0, flag = 0;

    poly_center.set(0, 0);
    for (int i = 0; i < 4; i++){
        for (int j = 0; j < 4; j++){
            flag = intersection(box_a_corners[i + 1], box_a_corners[i], box_b_corners[j + 1], box_b_corners[j], cross_points[cnt]);
            if (flag){
                poly_center = poly_center + cross_points[cnt];
                cnt++;
#ifdef DEBUG
                printf("Cross points (%.3f, %.3f): a(%.3f, %.3f)->(%.3f, %.3f), b(%.3f, %.3f)->(%.3f, %.3f) \n",
                    cross_points[cnt - 1].x, cross_points[cnt - 1].y,
                    box_a_corners[i].x, box_a_corners[i].y, box_a_corners[i + 1].x, box_a_corners[i + 1].y,
                    box_b_corners[i].x, box_b_corners[i].y, box_b_corners[i + 1].x, box_b_corners[i + 1].y);
#endif
            }
        }
    }

    // check corners
    for (int k = 0; k < 4; k++){
        if (check_in_box2d(box_a, box_b_corners[k])){
            poly_center = poly_center + box_b_corners[k];
            cross_points[cnt] = box_b_corners[k];
            cnt++;
#ifdef DEBUG
                printf("b corners in a: corner_b(%.3f, %.3f)", cross_points[cnt - 1].x, cross_points[cnt - 1].y);
#endif
        }
        if (check_in_box2d(box_b, box_a_corners[k])){
            poly_center = poly_center + box_a_corners[k];
            cross_points[cnt] = box_a_corners[k];
            cnt++;
#ifdef DEBUG
                printf("a corners in b: corner_a(%.3f, %.3f)", cross_points[cnt - 1].x, cross_points[cnt - 1].y);
#endif
        }
    }

    poly_center.y /= cnt;
    poly_center.z /= cnt;

    // sort the points of polygon
    Point temp;
    for (int j = 0; j < cnt - 1; j++){
        for (int i = 0; i < cnt - j - 1; i++){
            if (point_cmp(cross_points[i], cross_points[i + 1], poly_center)){
                temp = cross_points[i];
                cross_points[i] = cross_points[i + 1];
                cross_points[i + 1] = temp;
            }
        }
    }

#ifdef DEBUG
    printf("cnt=%d\n", cnt);
    for (int i = 0; i < cnt; i++){
        printf("All cross point %d: (%.3f, %.3f)\n", i, cross_points[i].x, cross_points[i].y);
    }
#endif

    // get the overlap areas
    float area = 0;
    for (int k = 0; k < cnt - 1; k++){
        area += cross(cross_points[k] - cross_points[0], cross_points[k + 1] - cross_points[0]);
    }

    return fabs(area) / 2.0;
}

__device__ inline float iou_bev(const float *box_a, const float *box_b){
    // params box_a: [x, y, z, dx, dy, dz, heading]
    // params box_b: [x, y, z, dx, dy, dz, heading]
    float sa = box_a[4] * box_a[5];
    float sb = box_b[4] * box_b[5];
    float s_overlap = box_overlap(box_a, box_b);
    return s_overlap / fmaxf(sa + sb - s_overlap, EPS);
}

__global__ void boxes_overlap_kernel(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_overlap){
    // params boxes_a: (N, 7) [x, y, z, dx, dy, dz, heading]
    // params boxes_b: (M, 7) [x, y, z, dx, dy, dz, heading]
    const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
    const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

    if (a_idx >= num_a || b_idx >= num_b){
        return;
    }
    const float * cur_box_a = boxes_a + a_idx * 7;
    const float * cur_box_b = boxes_b + b_idx * 7;
    float s_overlap = box_overlap(cur_box_a, cur_box_b);
    ans_overlap[a_idx * num_b + b_idx] = s_overlap;
}

__global__ void boxes_iou_bev_kernel(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_iou){
    // params boxes_a: (N, 7) [x, y, z, dx, dy, dz, heading]
    // params boxes_b: (M, 7) [x, y, z, dx, dy, dz, heading]
    const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
    const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

    if (a_idx >= num_a || b_idx >= num_b){
        return;
    }

    const float * cur_box_a = boxes_a + a_idx * 7;
    const float * cur_box_b = boxes_b + b_idx * 7;
    float cur_iou_bev = iou_bev(cur_box_a, cur_box_b);
    ans_iou[a_idx * num_b + b_idx] = cur_iou_bev;
}

__global__ void nms_kernel(const int boxes_num, const float nms_overlap_thresh,
                           const float *boxes, unsigned long long *mask){
    //params: boxes (N, 7) [x, y, z, dx, dy, dz, heading]
    //params: mask (N, N/THREADS_PER_BLOCK_NMS)

    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    // if (row_start > col_start) return;

    const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
    const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

    __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

    if (threadIdx.x < col_size) {
        block_boxes[threadIdx.x * 7 + 0] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 0];
        block_boxes[threadIdx.x * 7 + 1] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 1];
        block_boxes[threadIdx.x * 7 + 2] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 2];
        block_boxes[threadIdx.x * 7 + 3] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 3];
        block_boxes[threadIdx.x * 7 + 4] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 4];
        block_boxes[threadIdx.x * 7 + 5] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 5];
        block_boxes[threadIdx.x * 7 + 6] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 6];
    }
    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
        const float *cur_box = boxes + cur_box_idx * 7;

        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) {
          start = threadIdx.x + 1;
        }
        for (i = start; i < col_size; i++) {
            if (iou_bev(cur_box, block_boxes + i * 7) > nms_overlap_thresh){
                t |= 1ULL << i;
            }
        }
        const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
        mask[cur_box_idx * col_blocks + col_start] = t;
    }
}


__device__ inline float iou_normal(float const * const a, float const * const b) {
    //params: a: [x, y, z, dx, dy, dz, heading]
    //params: b: [x, y, z, dx, dy, dz, heading]

    float left = fmaxf(a[1] - a[4] / 2, b[1] - b[4] / 2), right = fminf(a[1] + a[4] / 2, b[1] + b[4] / 2);
    float top = fmaxf(a[2] - a[5] / 2, b[2] - b[5] / 2), bottom = fminf(a[2] + a[5] / 2, b[2] + b[5] / 2);
    float width = fmaxf(right - left, 0.f), height = fmaxf(bottom - top, 0.f);
    float interS = width * height;
    float Sa = a[4] * a[5];
    float Sb = b[4] * b[5];
    return interS / fmaxf(Sa + Sb - interS, EPS);
}


__global__ void nms_normal_kernel(const int boxes_num, const float nms_overlap_thresh,
                           const float *boxes, unsigned long long *mask){
    //params: boxes (N, 7) [x, y, z, dx, dy, dz, heading]
    //params: mask (N, N/THREADS_PER_BLOCK_NMS)

    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    // if (row_start > col_start) return;

    const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);
    const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS, THREADS_PER_BLOCK_NMS);

    __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

    if (threadIdx.x < col_size) {
        block_boxes[threadIdx.x * 7 + 0] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 0];
        block_boxes[threadIdx.x * 7 + 1] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 1];
        block_boxes[threadIdx.x * 7 + 2] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 2];
        block_boxes[threadIdx.x * 7 + 3] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 3];
        block_boxes[threadIdx.x * 7 + 4] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 4];
        block_boxes[threadIdx.x * 7 + 5] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 5];
        block_boxes[threadIdx.x * 7 + 6] = boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 7 + 6];
    }
    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
        const float *cur_box = boxes + cur_box_idx * 7;

        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) {
          start = threadIdx.x + 1;
        }
        for (i = start; i < col_size; i++) {
            if (iou_normal(cur_box, block_boxes + i * 7) > nms_overlap_thresh){
                t |= 1ULL << i;
            }
        }
        const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
        mask[cur_box_idx * col_blocks + col_start] = t;
    }
}





void boxesoverlapLauncher(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_overlap){

    dim3 blocks(DIVUP(num_b, THREADS_PER_BLOCK), DIVUP(num_a, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    boxes_overlap_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b, ans_overlap);
#ifdef DEBUG
    hipDeviceSynchronize();  // for using printf in kernel function
#endif
}

void boxesioubevLauncher(const int num_a, const float *boxes_a, const int num_b, const float *boxes_b, float *ans_iou){

    dim3 blocks(DIVUP(num_b, THREADS_PER_BLOCK), DIVUP(num_a, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    boxes_iou_bev_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b, ans_iou);
#ifdef DEBUG
    hipDeviceSynchronize();  // for using printf in kernel function
#endif
}


void nmsLauncher(const float *boxes, unsigned long long * mask, int boxes_num, float nms_overlap_thresh){
    dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
                DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
    dim3 threads(THREADS_PER_BLOCK_NMS);
    nms_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes, mask);
}


void nmsNormalLauncher(const float *boxes, unsigned long long * mask, int boxes_num, float nms_overlap_thresh){
    dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
                DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
    dim3 threads(THREADS_PER_BLOCK_NMS);
    nms_normal_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes, mask);
}
